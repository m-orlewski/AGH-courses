#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <assert.h>

float reduction_wrapper(float * input, unsigned int data_size);

inline hipError_t checkCudaError(hipError_t result);

float sumHost(const float * input, unsigned int data_size);

void init(float * input, unsigned int data_size);

__global__
void sumReduction(const float * input, int data_size, float * partial_sums)
{
	extern __shared__ float partSum[];
	
	int block_size = blockDim.x;
	int th = threadIdx.x;
	int start = 2 * blockIdx.x * block_size;

// Original shared memory initialization
	if (th+start < data_size)
		partSum[th] = input[start+th];
	else
		partSum[th] = 0;

	if (th+start+block_size < data_size)
		partSum[th+block_size] = input[start+th+block_size];
	else
		partSum[th+block_size] = 0;	

/*
//2nd shared memory optimization
	if (th+start < data_size)
	{
		if (th+start+block_size < data_size)
			partSum[th] = input[start+th] + input[start+th+block_size];
		else
			partSum[th] = input[start+th];
	}
	else
	{
		partSum[th] = 0;
	}
*/

//Original loop
	for (int stride=1; stride <= blockDim.x; stride *=2)
	{
		__syncthreads();
		if (th % stride == 0)
			partSum[2*th] += partSum[2*th + stride];
	}


/*
//1st loop optimization
	for (int i=block_size; i > 0; i /= 2)
	{
		__syncthreads();
		if (th < i)
			partSum[th] += partSum[th + i];
	}
/*

/*
//2nd loop optimization
	for (int i=block_size/2; i > 0; i /= 2)
	{
		__syncthreads();
		if (th < i)
			partSum[th] += partSum[th + i];
	}
*/
	
	__syncthreads();
	if (th == 0)
	{
		partial_sums[blockIdx.x] = partSum[0];
	} 
}

int main(int argc, char * argv[])
{
	unsigned int data_size;
	if (argc != 2)
	{
		fprintf(stderr, "Call to program should look like: %s <size of input data>\n", argv[0]);
		exit(EXIT_FAILURE);	
	}
	else
	{
		data_size = (unsigned int)atoi(argv[1]);
	}

	float * h_input = (float *)malloc(data_size * sizeof(float));
	if (h_input == NULL)
	{
		fprintf(stderr, "Unable to allocate memory for input array.\n");
		exit(EXIT_FAILURE);
	}

	init(h_input, data_size);

	float * d_input;
	checkCudaError(hipMalloc(&d_input, data_size * sizeof(float)));
	checkCudaError(hipMemcpy(d_input, h_input, data_size * sizeof(float), hipMemcpyHostToDevice));

	float device_sum = reduction_wrapper(d_input, data_size);
	float host_sum = sumHost(h_input, data_size);

	if (device_sum == host_sum)
	{
		printf("Results are correct! (%f == %f)\n", device_sum, host_sum);
	}
	else
	{
		printf("Something went wrong! (%f != %f)\n", device_sum, host_sum);
	}

	hipFree(d_input);
	free(h_input);

	return 0;
}

float reduction_wrapper(float * input, unsigned int data_size)
{
	int deviceId;
	checkCudaError(hipGetDevice(&deviceId));

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, deviceId);

	printf("\nDevice name: %s\n", prop.name);
	printf("Shared memory per block: %d\n", prop.sharedMemPerBlock);
	printf("Threads per block: %d\n", prop.maxThreadsPerBlock);
	printf("Number of SMs: %d\n", prop.multiProcessorCount);
	printf("Shared memory per SM: %d\n\n", prop.sharedMemPerMultiprocessor);

	int numOfThreads = prop.maxThreadsPerBlock;

	while (2 * numOfThreads * sizeof(float) > prop.sharedMemPerBlock)//to not excess our shared memory limit
	{
		numOfThreads -= 32;//to fully use warps
	}
	
	int partial_sums = data_size/(numOfThreads * 2); 

	if (data_size % (numOfThreads * 2))
	{
		partial_sums++;
	}

	int numOfBlocks = partial_sums; // each block will calculate one partial sum
	
	float * h_output = (float *)calloc(partial_sums, sizeof(float));
	if (h_output == NULL)
	{
		fprintf(stderr, "Unable to allocate memory for partial sums array.\n");
		exit(EXIT_FAILURE);
	}

	float * d_output;
	checkCudaError(hipMalloc(&d_output, partial_sums * sizeof(float)));
	checkCudaError(hipMemcpy(d_output, h_output, partial_sums * sizeof(float), hipMemcpyHostToDevice));

	hipEvent_t start, stop;
	checkCudaError(hipEventCreate(&start));
	checkCudaError(hipEventCreate(&stop));

	checkCudaError(hipEventRecord(start));

	sumReduction<<<numOfBlocks, numOfThreads, 2*numOfThreads*sizeof(float)>>>(input, data_size, d_output);
	checkCudaError(hipDeviceSynchronize());

	checkCudaError(hipMemcpy(h_output, d_output, partial_sums * sizeof(float), hipMemcpyDeviceToHost));

	float sum = 0;
	for (int i=0; i < partial_sums; i++)
	{	
		sum += h_output[i];
	}
	checkCudaError(hipEventRecord(stop));

	float time = 0;
	checkCudaError(hipEventElapsedTime(&time, start, stop));

	printf("Sum on GPU was calculated in %f miliseconds\n", time);

	hipFree(d_output);
	free(h_output);

	return sum;
}

inline hipError_t checkCudaError(hipError_t result)
{
	if (result != hipSuccess)
	{
		fprintf(stderr, "Cuda Error: %s\n", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
	return result;
}

float sumHost(const float * input, unsigned int data_size)
{
	clock_t start, stop;

	start = clock();	

	float sum = 0;
	for (unsigned int i=0; i < data_size; i++)
		sum += input[i];

	stop = clock();

	printf("Sum on CPU was calculated in %f miliseconds\n", 1000 * (double)(stop - start)/(double)CLOCKS_PER_SEC);

	return sum;
}

void init(float * input, unsigned int data_size)
{
	for (unsigned int i=0; i < data_size; i++)
	{
		input[i] = (float)(rand() % 2);
	}	
} 
