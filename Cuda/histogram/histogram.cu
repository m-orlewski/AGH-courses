
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <hip/hip_runtime.h>

const unsigned int MAX_BINS=4096;
const unsigned int SATURATION=256;

//wrapper function launches thathistogram kernel,
void histogram_wrapper(const unsigned int * input, const unsigned int data_size, unsigned int * histogram, hipEvent_t start, hipEvent_t stop);

//checks CUDA function calls for errors
inline hipError_t checkCudaError(hipError_t result);

//initializes input data
void init(unsigned int * input, const unsigned int data_size);

//histogram function
void histogram_function(const unsigned int * input, const unsigned int data_size, unsigned int * histogram);

//histogram kernel
__global__
void histogram_kernel(const unsigned int * input, const unsigned int data_size, unsigned int * histogram)
{
	extern __shared__ unsigned int local_histogram[];
	unsigned int th = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int stride = gridDim.x * blockDim.x;

	// initializing local_histogram
	for (unsigned int i=threadIdx.x; i < MAX_BINS; i += blockDim.x)
	{
		local_histogram[i] = 0;
	}

	__syncthreads();

	// filling local histogram
	for (unsigned int i=th; i < data_size; i += stride)
	{
		atomicAdd(&local_histogram[input[i]], 1);
	}

	__syncthreads();

	//updating global histogram
	for (unsigned int i=threadIdx.x; i < MAX_BINS; i += blockDim.x)
	{
		atomicAdd(&histogram[i], local_histogram[i]);
	}

}

//saturation kernel
__global__
void saturation(unsigned int * histogram)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i=tid; i < MAX_BINS; i+=stride)
	{
		if (histogram[i]>SATURATION)
		{
			histogram[i] = SATURATION;
		}
	}
}


int main(int argc, char * argv[])
{
	unsigned int data_size;
	if (argc != 2)
	{
		fprintf(stderr, "Call to program should look like: %s <size of input data>\n", argv[0]);
		exit(EXIT_FAILURE);
	}
	else
	{
		data_size = (unsigned int)atoi(argv[1]);
	}

	//Host memory allocation
	unsigned int * h_input = (unsigned int *)malloc(data_size * sizeof(unsigned int));
	unsigned int * h_histogram = (unsigned int *)calloc(MAX_BINS, sizeof(unsigned int));

	if (!h_input || !h_histogram)
	{
		fprintf(stderr, "Memory allocation error.\n");
		exit(EXIT_FAILURE);
	}

	//Device memory allocation
	unsigned int * d_input;
	checkCudaError(hipMalloc(&d_input, data_size*sizeof(unsigned int)));	
	
	unsigned int * d_histogram;
	checkCudaError(hipMalloc(&d_histogram, MAX_BINS*sizeof(unsigned int)));

	// Creating events for timing histogram kernel
	hipEvent_t start, stop;
	checkCudaError(hipEventCreate(&start));
	checkCudaError(hipEventCreate(&stop));

	// Coping memory from host to device
	checkCudaError(hipMemcpy(d_input, h_input, data_size*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaError(hipMemcpy(d_histogram, h_histogram, MAX_BINS*sizeof(unsigned int), hipMemcpyHostToDevice));

	histogram_wrapper(d_input, data_size, d_histogram, start, stop);

	// Coping memory from device to host
	checkCudaError(hipMemcpy(h_input, d_input, data_size*sizeof(unsigned int), hipMemcpyDeviceToHost));
	checkCudaError(hipMemcpy(h_histogram, d_histogram, MAX_BINS*sizeof(unsigned int), hipMemcpyDeviceToHost));
	
	unsigned int * h_histogram2 = (unsigned int *)calloc(MAX_BINS, sizeof(int));
	histogram_function(h_input, data_size, h_histogram2);
	

	for (int i=0; i < MAX_BINS; i++)
	{
		if (h_histogram[i] != h_histogram2[i])
		{
			printf("Histogram results do not match!\n");
			exit(EXIT_FAILURE);
		}
	}

	printf("Everything is fine!\n");

	

	return 0;
}

void histogram_wrapper(const unsigned int * input, const unsigned int data_size, unsigned int * histogram, hipEvent_t start, hipEvent_t stop)
{
	int deviceId;
	checkCudaError(hipGetDevice(&deviceId));

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, deviceId);

	printf("\nDevice name: %s\n", prop.name);
	printf("Shared memory per block: %d\n", prop.sharedMemPerBlock);
	printf("Threads per block: %d\n", prop.maxThreadsPerBlock);
	printf("Number of SMs: %d\n", prop.multiProcessorCount);
	printf("Shared memory per SM: %d\n\n", prop.sharedMemPerMultiprocessor);

	checkCudaError(hipEventRecord(start));
	histogram_kernel<<<prop.multiProcessorCount, prop.maxThreadsPerBlock, MAX_BINS*sizeof(unsigned int)>>>(input, data_size, histogram);
	checkCudaError(hipEventRecord(stop));
	checkCudaError(hipDeviceSynchronize());

	checkCudaError(hipEventSynchronize(stop));
	float time = 0;
	checkCudaError(hipEventElapsedTime(&time, start, stop));

	printf("Kernel was executed in %f miliseconds\n", time);

	//Saturation
	saturation<<<16, 256>>>(histogram); //pre set for 4096 bins

	checkCudaError(hipDeviceSynchronize());
}

inline hipError_t checkCudaError(hipError_t result)
{
        if (result != hipSuccess)
        {
                fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(result));
                assert(result == hipSuccess);
        }
        return result;
}

void init(unsigned int * input, const unsigned int data_size)
{
	for (unsigned int i=0; i < data_size; i++)
	{
		input[i] = (unsigned int)rand()%MAX_BINS;
	}
}

void histogram_function(const unsigned int * input, const unsigned int data_size, unsigned int * histogram)
{
	double start, stop;
	
	start = clock();
	for (unsigned int i=0; i < data_size; i++)
	{
		histogram[input[i]]++;
	}
	stop = clock();

	printf("Function was executed in %lf miliseconds\n",1000.0* (stop-start)/(double)CLOCKS_PER_SEC);

	//Saturation
	for (int i=0; i < MAX_BINS; i++)
	{
		if (histogram[i] > SATURATION)
		{
			histogram[i] = SATURATION;
		}
	}
}
